#include "hip/hip_runtime.h"
#include "my_cuda_utils.hpp"

constexpr int BLOCKSIZE = 512;

template <unsigned int blockSize>
__device__ void warpReduce(volatile int *sdata, unsigned int tid) {
   if (blockSize >= 64) {
      sdata[tid] += sdata[tid + 32];
   }
   if (blockSize >= 32) { 
      sdata[tid] += sdata[tid + 16];
   }
   if (blockSize >= 16) {
      sdata[tid] += sdata[tid + 8];
   }
   if (blockSize >= 8) {
      sdata[tid] += sdata[tid + 4];
   }
   if (blockSize >= 4) {
      sdata[tid] += sdata[tid + 2];
   }
   if (blockSize >= 2) {
      sdata[tid] += sdata[tid + 1];
   }
}

// Must be called twice. The second time with 1 block and 'num_blocks' threads
template <unsigned int blockSize>
__global__ void my_reduce(int* g_odata, int* g_idata, unsigned int n) {

   extern __shared__ int sdata[];
   unsigned int tid = threadIdx.x;
   unsigned int i = blockIdx.x*(blockSize*2) + tid;
   unsigned int gridSize = blockSize*2*gridDim.x;

   sdata[tid] = 0;
   __syncthreads();

   while (i < n) { 
      sdata[tid] += g_idata[i] + g_idata[i+blockSize]; 
      i += gridSize; 
   }
   __syncthreads();

   if (blockSize >= 512) { 
      if (tid < 256) { 
         sdata[tid] += sdata[tid + 256]; 
      } 
      __syncthreads(); 
   }
   
   if (blockSize >= 256) {
      if (tid < 128) {
      sdata[tid] += sdata[tid + 128]; 
      } 
      __syncthreads(); 
   }

   if (blockSize >= 128) { 
      if (tid < 64) {
      sdata[tid] += sdata[tid + 64]; 
      } 
      __syncthreads(); 
   }
   
   if (tid < 32) {
      warpReduce<blockSize>(sdata, tid);
   }
   __syncthreads(); 

   if (tid == 0) {
      g_odata[blockIdx.x] = sdata[0];
   }
}

// SINGLE PASS REDUCTION (Only if the XOR operator is supported natively by 
// an atomic operator in HW). The output must be initialized to zero before calling
// this kernel! The kernel cannot do this because 'CUDA's execution model does 
// not enable the race condition to be resolved between thread blocks'
// From CUDA Handbook, A Comprehensive Guide to GPU Programming by Nicholas Wilt
// (WOW!)
__global__ void reduce_with_atomic( int *out, const int *in, size_t N ) {
   const int tid = threadIdx.x; 
   int partialSum = 0; 
   size_t i = blockIdx.x*blockDim.x + tid;;
   for ( ; i < N; i += blockDim.x*gridDim.x ) {
      partialSum += in[i]; 
   } 
   atomicAdd( &out[i], partialSum ); 
}



int main( int argc, char* argv[] ) {
   try {
      hipError_t cerror = hipSuccess;
      bool debug = false;
      int num_vals = 2050;
      int* sums = nullptr;
      int* vals = nullptr;
      int* d_sums = nullptr;
      int* d_vals = nullptr;

      size_t num_bytes = num_vals * sizeof( int );
      
      int device_id = -1;
      try_cuda_func_throw( cerror, hipGetDevice( &device_id ) );

      std::unique_ptr<hipStream_t> stream_ptr = my_make_unique<hipStream_t>();
      try_cudaStreamCreate( stream_ptr.get() );
      
      try_cuda_func_throw( cerror, hipHostAlloc( (void**)&sums, num_bytes, hipHostMallocDefault ) );
      try_cuda_func_throw( cerror, hipHostAlloc( (void**)&vals, num_bytes, hipHostMallocDefault ) );

      for( int index = 0; index < num_vals; index++ ) {
         sums[index] = 0;
         vals[index] = index+1;
         if ( debug ) {
            if ((index < 10) || (index > (num_vals-10))) {
               printf("vals[%d] = %d\n", index, vals[index] );
            }
            if ( index == 11 ) {
               printf( "...\n" ); 
            }
         }
      }
      printf( "\n" ); 

      try_cuda_func_throw( cerror, hipMalloc( (void**)&d_sums, num_bytes ) );
      try_cuda_func_throw( cerror, hipMalloc( (void**)&d_vals, num_bytes ) );

      int exp_sum = (num_vals*(num_vals+1))/2;

      int threads_per_block = BLOCKSIZE;
      int num_blocks = (num_vals + threads_per_block - 1) / threads_per_block;

      if ( debug ) {
         printf( "num_vals = %d\n", num_vals );
         printf( "num_blocks = %d\n", num_blocks );
         printf( "threads_per_block is = %d\n", threads_per_block );
         printf( "BLOCKSIZE is = %d\n", BLOCKSIZE );
         printf( "actual number of threads will be %d\n\n", (num_blocks * threads_per_block) ); 
      }

      ///////////////////////////////////////////////
      // TWO PASS REDUCE
      //////////////////////////////////////////////////
      size_t num_shared_bytes = threads_per_block * sizeof(int);
      
      Time_Point start = Steady_Clock::now();

      try_cuda_func_throw( cerror, hipMemcpyAsync( d_vals, vals, num_bytes,
               hipMemcpyHostToDevice, *(stream_ptr.get()) ) );
      
      my_reduce<BLOCKSIZE><<<num_blocks, threads_per_block, num_shared_bytes, *(stream_ptr.get())>>>( d_sums, d_vals, num_vals );
      //try_cuda_func_throw( cerror, hipPeekAtLastError() );
      my_reduce<1><<<1, threads_per_block, num_shared_bytes, *(stream_ptr.get())>>>( d_sums, d_sums, num_vals );

      try_cuda_func_throw( cerror, hipMemcpyAsync( sums, d_sums, num_bytes,
               hipMemcpyDeviceToHost, *(stream_ptr.get()) ) );
      
      try_cuda_func_throw( cerror, hipDeviceSynchronize() );

      Time_Point stop = Steady_Clock::now();
      Duration_ms duration_ms = stop - start;

      if ( debug ) {
         for( int index = 0; index < num_vals; ++index ) {
            printf("After GPU: Sum %d is %d\n", index, sums[index] );
         } 
         printf("\n\n");
         printf("Sum is %d\n", sums[0] );
         printf("Expected Sum is %d\n\n", exp_sum );
      }

      if ( sums[0] != exp_sum ) {
         throw std::runtime_error( std::string{ __func__ } + std::string{"(): MISMATCH: two pass reduce: expected sum = "} +
            std::to_string(exp_sum) + std::string{", actual sum = "} + std::to_string( sums[0] )  );
      }
      float milliseconds = duration_ms.count();
      printf( "Two pass reduce with shared memory: All results matched expected. It took %f milliseconds to reduce %d values\n\n", milliseconds, num_vals );


      ///////////////////////////////////////////////
      // REDUCE WITH ATOMIC ADD
      //////////////////////////////////////////////////

      // Clear the sums from the previous run
      try_cuda_func_throw( cerror, hipMemset( sums, 0, sizeof(int) ) );
      for( int index = 0; index < num_vals; ++index ) {
         sums[index] = 0;
      } 
      num_shared_bytes = 0u;

      if ( debug ) printf("Before reduce_with_atomic, sum is %d\n", sums[0] );
   
      start = Steady_Clock::now();
      
      try_cuda_func( cerror, hipMemcpyAsync( d_vals, vals, num_bytes,
               hipMemcpyHostToDevice, *(stream_ptr.get()) ) );
      
      reduce_with_atomic<<< num_blocks, threads_per_block, num_shared_bytes, *(stream_ptr.get())>>>( d_sums, d_vals, num_vals );
      try_cuda_func_throw( cerror, hipPeekAtLastError() );
      
      try_cuda_func_throw( cerror, hipMemcpyAsync( sums, d_sums, num_bytes,
               hipMemcpyDeviceToHost, *(stream_ptr.get()) ) );

      try_cuda_func_throw( cerror, hipDeviceSynchronize() );
      stop = Steady_Clock::now();
      duration_ms = stop - start;

      if ( debug ) {
         printf("Sum from reduce with atomicAdd() is %d\n", sums[0] );
         printf("Expected Sum is %d\n\n", exp_sum );
      }
      if ( sums[0] != exp_sum ) {
         throw std::runtime_error( std::string{ __func__ } + std::string{"(): MISMATCH: reduce with atomicAdd(): expected sum = "} +
            std::to_string(exp_sum) + std::string{", actual sum = "} + std::to_string( sums[0] )  );
      }
      milliseconds = duration_ms.count();
      printf( "Single pass reduce with atomicAdd(): All results matched expected. It took %f milliseconds to reduce %d values\n\n", milliseconds, num_vals );

      return EXIT_SUCCESS;

   } catch( std::exception& ex ) {
      std::cout << __func__ << "(): ERROR: " << ex.what() << "\n"; 
      return EXIT_FAILURE;
   }
}
